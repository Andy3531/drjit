/*
    src/cuda/common.cpp -- CUDA backend (wrapper routines)

    Enoki is a C++ template library that enables transparent vectorization
    of numerical kernels using SIMD instruction sets available on current
    processor architectures.

    Copyrighe (c) 2018 Wenzel Jakob <wenzel.jakob@epfl.ch>

    All rights reserved. Use of this source code is governed by a BSD-style
    license that can be found in the LICENSE file.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "common.cuh"

NAMESPACE_BEGIN(enoki)

ENOKI_EXPORT void* cuda_malloc(size_t size) {
    void *result = nullptr;
    cuda_check(hipMalloc(&result, size));
    return result;
}

ENOKI_EXPORT void* cuda_malloc_zero(size_t size) {
    void *result = nullptr;
    cuda_check(hipMalloc(&result, size));
    cuda_check(hipMemsetAsync(result, 0, size));
    return result;
}

ENOKI_EXPORT void* cuda_managed_malloc(size_t size) {
    void *result = nullptr;
    cuda_check(hipMallocManaged(&result, size));
    return result;
}

ENOKI_EXPORT void cuda_free(void *ptr) {
    cuda_check(hipFree(ptr));
}

ENOKI_EXPORT void cuda_memcpy_to_device(void *dst, const void *src, size_t size) {
    cuda_check(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

ENOKI_EXPORT void cuda_memcpy_from_device(void *dst, const void *src, size_t size) {
    cuda_check(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
}

NAMESPACE_END(enoki)
